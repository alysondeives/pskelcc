#include "hip/hip_runtime.h"
/* Available optimizations (value should be used as the first parameter in the command line):
   0 (o0) -> no optimization
   1 (o1) -> shared memory
   2 (o2) -> for iteration on Z axis (Paulius)
   3 (o3) -> for iteration on Z axis without using registers
   25 (o2_o5) -> for iteration on Z axis (Paulius) + pragma unroll
   12 (o1_o2) -> shared memory + for iteration on Z axis
   13 (o1_o3) -> shared memory + for iteration on Z axis without registers
   124 (o1_o2_o4) -> shared memory + for iteration on Z axis + temporal blocking
   125 (o1_o2_o5) -> shared memory + for iteration on Z axis + pragma unroll
   7 (o7) -> use of read only cache (__restrict__ and const modifiers)
   17 (o1_o7) -> use of shared memory + read only cache (__restrict__ and const modifiers)
   27 (o2_o7) -> for iteration on Z axis + read only cache
   37 (o3_o7) -> for iteration on Z axis without registers + read only cache
   1247 (o1_o2_o4_o7) -> shared memory + read only cache + for iteration on Z axis + temporal blocking

   Known limitations: data matrix size must be multiple of BLOCK_SIZE
*/

#include <stdio.h>

//#define PRINT_GOLD
//#define PRINT_RESULT

#define BLOCK_DIMX 32
#define BLOCK_DIMY 16
#define BLOCK_DIMZ 1
#define RADIUS 1 // Half of the order

// Uncomment the following line to enable inside kernel run time measurement
//#define MEASURE_KERNEL_RUNTIME

// Error checking function
#define wbCheck(stmt) do {                                                    \
        hipError_t err = stmt;                                               \
        if (err != hipSuccess) {                                             \
            printf("ERROR: Failed to run stmt %s\n", #stmt);                       \
            printf("ERROR: Got CUDA error ...  %s\n", hipGetErrorString(err));    \
            return -1;                                                        \
        }                                                                     \
    } while(0)

// Elapsed clock cycles read function
#ifdef MEASURE_KERNEL_RUNTIME
#define read_clk_cycles(var,tindex) if ( (row == tindex) && (col == tindex) && (depth == tindex) ) { \
                                       var = clock64();						     \
                                    }
#else
#define read_clk_cycles(var,tindex)
#endif


__constant__ float coeff[RADIUS*6+1];

/* 
   Optimization o0: baseline code (no optimization)
*/
__global__ void calc_stencil_o0(float *a, float *b, int dimx, int dimy) {

  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
	
  int row = blockIdx.y * blockDim.y + ty;
  int col = blockIdx.x * blockDim.x + tx;
  int depth = blockIdx.z * blockDim.z + tz;
	
  // (depth/row/col + 1) to compensate the halo
  int index = (depth+1) * dimx * dimy + (row+1) * dimx + (col+1);
  
  // Compute stencil
  b[index] = coeff[0] * a[index] +
    coeff[1] * a[index-1] +
    coeff[2] * a[index+1] +
    coeff[3] * a[index-dimx] +
    coeff[4] * a[index+dimx] +
    coeff[5] * a[index+(dimx*dimy)] +
    coeff[6] * a[index-(dimx*dimy)];
}

/* 
   Optimization o0: baseline code (no optimization), without using constant coeff
*/
__global__ void calc_stencil_o0_coeff(float *a, float *b, int dimx, int dimy, float c0, float c1, float c2, float c3, float c4, float c5, float c6) {

  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
	
  int row = blockIdx.y * blockDim.y + ty;
  int col = blockIdx.x * blockDim.x + tx;
  int depth = blockIdx.z * blockDim.z + tz;
	
  // (depth/row/col + 1) to compensate the halo
  int index = (depth+1) * dimx * dimy + (row+1) * dimx + (col+1);
  
  // Compute stencil
  b[index] = c0 * a[index] +
    c1 * a[index-1] +
    c2 * a[index+1] +
    c3 * a[index-dimx] +
    c4 * a[index+dimx] +
    c5 * a[index+(dimx*dimy)] +
    c6 * a[index-(dimx*dimy)];
}

/* 
   Optimization o1: shared memory
*/
__global__ void calc_stencil_o1(float *a, float *b, int dimx, int dimy, clock_t *runtime) {

  // Shared Memory Declaration
  __shared__ float ds_a[BLOCK_DIMY+2*RADIUS][BLOCK_DIMX+2*RADIUS];

  int tx = threadIdx.x + RADIUS;
  int ty = threadIdx.y + RADIUS;
  int tz = threadIdx.z + RADIUS;
	
  int row = blockIdx.y * blockDim.y + ty;
  int col = blockIdx.x * blockDim.x + tx;
  int depth = blockIdx.z * blockDim.z + tz;
  
  int index = (depth) * dimx * dimy + (row) * dimx + (col);

  int stride = dimx * dimy; // Distance between 2D slices

  // Measure clock() init of every thread
  // runtime[(depth-RADIUS) * (dimx-2*RADIUS) * (dimy-2*RADIUS) + (row-RADIUS) * (dimx-2*RADIUS) + (col-RADIUS)] = clock64();

  // read_clk_cycles(runtime[0],RADIUS);

  // Load above/below halo data to shared memory
  if (threadIdx.y < RADIUS) {
    ds_a[threadIdx.y][tx] = a[index-(RADIUS*dimx)];
    ds_a[threadIdx.y + BLOCK_DIMY + RADIUS][tx] = a[index+(BLOCK_DIMY*dimx)];
  }

  // Load left/right halo data to shared memory
  if (threadIdx.x < RADIUS) {
    ds_a[ty][threadIdx.x] = a[index-RADIUS];
    ds_a[ty][threadIdx.x + BLOCK_DIMX + RADIUS] = a[index+BLOCK_DIMX];
  }

  // Load current position to shared memory
  ds_a[ty][tx] = a[index];

  __syncthreads();

  // read_clk_cycles(runtime[1],RADIUS);

  // Compute stencil
  b[index] = coeff[0] * ds_a[ty][tx] +
    coeff[1] * ds_a[ty][tx-1] +
    coeff[2] * ds_a[ty][tx+1] +
    coeff[3] * ds_a[ty-1][tx] +
    coeff[4] * ds_a[ty+1][tx] +
    coeff[5] * a[index-stride] +
    coeff[6] * a[index+stride];

  // read_clk_cycles(runtime[2],RADIUS);
}

/* 
   Optimization o2: for iteration on Z axis
*/
__global__ void calc_stencil_o2(float *a, float *b, int dimx, int dimy, int dimz) {

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
	
  // (depth/row/col + RADIUS) to compensate the halo
  int in_index = (row+RADIUS) * dimx + (col+RADIUS); // Index for reading Z values (+RADIUS to compensate the halo zones)
  int out_index = 0; // Index for writing output
  
  int stride = dimx * dimy; // Distance between 2D slices
  
  register float infront; // Variable to store the value in front (in the Z axis) of the current slice
  register float behind; // Variable to store the value behind (in the Z axis) the current slice
  register float current; // Input value in the current slice

  // Load initial values (behind will be loaded inside the next 'for')
  current = a[in_index];
  out_index = in_index;
  in_index += stride;
  
  infront = a[in_index];
  in_index += stride;

  // Iterate over the Z axis
  for (int i = RADIUS; i < dimz - RADIUS; i++) {

    // Load the new values in Z axis
    behind = current;
    current = infront;
    infront = a[in_index];

    in_index += stride;
    out_index += stride;

    // Compute stencil
    b[out_index] = coeff[0] * current +
      coeff[1] * a[out_index-1] +
      coeff[2] * a[out_index+1] +
      coeff[3] * a[out_index-dimx] +
      coeff[4] * a[out_index+dimx] +
      coeff[5] * behind +
      coeff[6] * infront;
  }

}

/* 
   Optimization o3: for iteration on Z axis without using registers
*/
__global__ void calc_stencil_o3(float *a, float *b, int dimx, int dimy, int dimz) {

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
	
  // (depth/row/col + RADIUS) to compensate the halo
  int out_index = (row+RADIUS) * dimx + (col+RADIUS); // Index for writing output
  
  int stride = dimx * dimy; // Distance between 2D slices

  // Iterate over the Z axis
  for (int i = RADIUS; i < dimz - RADIUS; i++) {

    out_index += stride;

    // Compute stencil
    b[out_index] = coeff[0] * a[out_index] +
      coeff[1] * a[out_index-1] +
      coeff[2] * a[out_index+1] +
      coeff[3] * a[out_index-dimx] +
      coeff[4] * a[out_index+dimx] +
      coeff[5] * a[out_index-stride] +
      coeff[6] * a[out_index+stride];
  }

}

/* 
   Optimization o2_o5: for iteration on Z axis + pragma unroll
*/
__global__ void calc_stencil_o2_o5(float *a, float *b, int dimx, int dimy, int dimz) {

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
	
  // (depth/row/col + RADIUS) to compensate the halo
  int in_index = (row+RADIUS) * dimx + (col+RADIUS); // Index for reading Z values (+RADIUS to compensate the halo zones)
  int out_index = 0; // Index for writing output
  
  int stride = dimx * dimy; // Distance between 2D slices
  
  register float infront; // Variable to store the value in front (in the Z axis) of the current slice
  register float behind; // Variable to store the value behind (in the Z axis) the current slice
  register float current; // Input value in the current slice

  // Load initial values (behind will be loaded inside the next 'for')
  current = a[in_index];
  out_index = in_index;
  in_index += stride;
  
  infront = a[in_index];
  in_index += stride;

  // Iterate over the Z axis
  #pragma unroll
  for (int i = RADIUS; i < dimz - RADIUS; i++) {

    // Load the new values in Z axis
    behind = current;
    current = infront;
    infront = a[in_index];

    in_index += stride;
    out_index += stride;

    // Compute stencil
    b[out_index] = coeff[0] * current +
      coeff[1] * a[out_index-1] +
      coeff[2] * a[out_index+1] +
      coeff[3] * a[out_index-dimx] +
      coeff[4] * a[out_index+dimx] +
      coeff[5] * behind +
      coeff[6] * infront;
  }

}

/* 
   Optimization o1_o2: for iteration on Z axis + use of shared memory
*/
__global__ void calc_stencil_o1_o2(float *a, float *b, int dimx, int dimy, int dimz, clock_t *runtime) {

  // Shared memory declaration
  __shared__ float ds_a[BLOCK_DIMY+2*RADIUS][BLOCK_DIMX+2*RADIUS];

  int tx = threadIdx.x + RADIUS;
  int ty = threadIdx.y + RADIUS;

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
	
  // (depth/row/col + RADIUS) to compensate the halo
  int in_index = (row+RADIUS) * dimx + (col+RADIUS); // Index for reading Z values (+1 to compensate the halo zones)
  int out_index = 0; // Index for writing output
  
  int stride = dimx * dimy; // Distance between 2D slices
  
  register float infront; // Variable to store the value in front (in the Z axis) of the current slice
  register float behind; // Variable to store the value behind (in the Z axis) the current slice
  register float current; // Input value in the current slice

  // register clock_t time_aux1 = 0, time_aux2 = 0, time_aux3 = 0, time_aux4 = 0, time_aux5 = 0;

  // Load initial values (behind will be loaded inside the next 'for')
  current = a[in_index];
  out_index = in_index;
  in_index += stride;
  
  infront = a[in_index];
  in_index += stride;

  // Iterate over the Z axis
  for (int i = RADIUS; i < dimz - RADIUS; i++) {

    // Load the new values in Z axis
    behind = current;
    current = infront;
    infront = a[in_index];

    in_index += stride;
    out_index += stride;

    // if ( (row == RADIUS) && (col == RADIUS) ) { 
    //   time_aux1 = clock64();
    // }

    // Load above/below halo data to shared memory
    if (threadIdx.y < RADIUS) {
      ds_a[threadIdx.y][tx] = a[out_index - (RADIUS * dimx)];
      ds_a[threadIdx.y + BLOCK_DIMY + RADIUS][tx] = a[out_index + (dimx * BLOCK_DIMY)];
    }

    // Load left/right halo data to shared memory
    if (threadIdx.x < RADIUS) {
      ds_a[ty][threadIdx.x] = a[out_index - RADIUS];
      ds_a[ty][threadIdx.x + BLOCK_DIMX + RADIUS] = a[out_index + BLOCK_DIMX];
    }

    // Load current position to shared memory
    ds_a[ty][tx] = current;

    __syncthreads();

    // if ( (row == RADIUS) && (col == RADIUS) ) {
    //   time_aux2 = clock64();
    //   time_aux4 += time_aux2 - time_aux1;
    // }

    // Compute stencil (7 single precision mul + 6 single precision add)
    b[out_index] = coeff[0] * current +
      coeff[1] * ds_a[ty][tx-1] +
      coeff[2] * ds_a[ty][tx+1] +
      coeff[3] * ds_a[ty-1][tx] +
      coeff[4] * ds_a[ty+1][tx] +
      coeff[5] * behind +
      coeff[6] * infront;

    __syncthreads();
  
    // if ( (row == RADIUS) && (col == RADIUS) ) {
    //   time_aux3 = clock64();
    //   time_aux5 += time_aux3 - time_aux2;
    // }
  }
  // if ( (row == RADIUS) && (col == RADIUS) ) {
  //   runtime[0] = time_aux4;
  //   runtime[1] = time_aux5;
  // }
}

/* 
   Optimization o1_o3: for iteration on Z axis without registers + use of shared memory
*/
__global__ void calc_stencil_o1_o3(float *a, float *b, int dimx, int dimy, int dimz, clock_t *runtime) {

  // Shared memory declaration
  __shared__ float ds_a[BLOCK_DIMY+2*RADIUS][BLOCK_DIMX+2*RADIUS];

  int tx = threadIdx.x + RADIUS;
  int ty = threadIdx.y + RADIUS;

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
	
  // (depth/row/col + RADIUS) to compensate the halo
  int out_index = (row+RADIUS) * dimx + (col+RADIUS); // Index for writing output
  
  int stride = dimx * dimy; // Distance between 2D slices
  
  // Iterate over the Z axis
  for (int i = RADIUS; i < dimz - RADIUS; i++) {

    out_index += stride;

    // Load above/below halo data to shared memory
    if (threadIdx.y < RADIUS) {
      ds_a[threadIdx.y][tx] = a[out_index - (RADIUS * dimx)];
      ds_a[threadIdx.y + BLOCK_DIMY + RADIUS][tx] = a[out_index + (dimx * BLOCK_DIMY)];
    }

    // Load left/right halo data to shared memory
    if (threadIdx.x < RADIUS) {
      ds_a[ty][threadIdx.x] = a[out_index - RADIUS];
      ds_a[ty][threadIdx.x + BLOCK_DIMX + RADIUS] = a[out_index + BLOCK_DIMX];
    }

    // Load current position to shared memory
    ds_a[ty][tx] = a[out_index];

    __syncthreads();

    // Compute stencil (7 single precision mul + 6 single precision add)
    b[out_index] = coeff[0] * ds_a[ty][tx] +
      coeff[1] * ds_a[ty][tx-1] +
      coeff[2] * ds_a[ty][tx+1] +
      coeff[3] * ds_a[ty-1][tx] +
      coeff[4] * ds_a[ty+1][tx] +
      coeff[5] * a[out_index-stride] +
      coeff[6] * a[out_index+stride]; 

    __syncthreads();
  }
}

/* 
   Optimization o1_o2_o4: shared memory + for iteration on Z axis + temporal blocking (will always compute 2 time iterations)
*/
__global__ void calc_stencil_o1_o2_o4(float *a, float *b, int dimx, int dimy, int dimz) {

  // Shared memory declaration
  __shared__ float ds_a[BLOCK_DIMY+2*RADIUS][BLOCK_DIMX+2*RADIUS][2];

  int tx = threadIdx.x + RADIUS;
  int ty = threadIdx.y + RADIUS;

  int row = blockIdx.y * (BLOCK_DIMY-2*RADIUS) + threadIdx.y + RADIUS;
  int col = blockIdx.x * (BLOCK_DIMX-2*RADIUS) + threadIdx.x + RADIUS;
	
  int in_index = row * dimx + col; // Index for reading Z values
  int out_index = 0; // Index for writing output
  int next_index = 0; // Index for plane Z = output + RADIUS
  
  int stride = dimx * dimy; // Distance between 2D slices

  // t0 = t + 0
  register float t0_infront1; // Variable to store the value ahead (in the Z axis) of the current slice
  register float t0_behind1; // Variable to store the value behind (in the Z axis) the current slice
  register float t0_current; // Input value in the current slice

  // t1 = t + 1
  register float t1_infront1; // Variable to store the value ahead (in the Z axis) of the current slice
  register float t1_behind1; // Variable to store the value behind (in the Z axis) the current slice
  register float t1_current; // Value in current slice for t+1

  // Load ghost zones
  in_index += stride;
  t0_behind1 = a[in_index]; // Z = -R = -1
  in_index += stride;
  next_index = in_index; // Z = R-1 = 0
  
  out_index = in_index; // Index for writing output, Z = 0
  
  t0_current = a[in_index]; // Z = 0
  in_index += stride;
  t0_infront1 = a[in_index]; // Z = (2R-1) = 1
  in_index += stride;

  // Load Z = 0 to shared memory
  // Load above/below halo data
  if (threadIdx.y < RADIUS) {
    ds_a[threadIdx.y][tx][1] = a[out_index - (RADIUS * dimx)];
    ds_a[threadIdx.y + BLOCK_DIMY + RADIUS][tx][1] = a[out_index + (dimx * BLOCK_DIMY)];
  }
  
  // Load left/right halo data
  if (threadIdx.x < RADIUS) {
    ds_a[ty][threadIdx.x][1] = a[out_index - RADIUS];
    ds_a[ty][threadIdx.x + BLOCK_DIMX + RADIUS][1] = a[out_index + BLOCK_DIMX];
  }
  ds_a[ty][tx][1] = t0_current;

  __syncthreads();

  // Compute stencil for Z = 0 (t + 1) but exclude ghost zones 
   if ( (row >= 2*RADIUS) && (row < (dimy-2*RADIUS)) && (col >= 2*RADIUS) && (col < (dimx-2*RADIUS)) ) {
    t1_current = coeff[0] * t0_current +
     coeff[1] * ds_a[ty][tx-1][1] +
      coeff[2] * ds_a[ty][tx+1][1] +
      coeff[3] * ds_a[ty-1][tx][1] +
      coeff[4] * ds_a[ty+1][tx][1] +
      coeff[5] * t0_behind1 +
      coeff[6] * t0_infront1;
   } else {
     t1_current = t0_current;
   }
  
  // Copy planes Z = -1 to -R to registers in t+1 (ghost zones, keep values in 0.0)
  t1_behind1 = t0_behind1;
  
  __syncthreads();

  for (int i = 0; i < dimz-(4*RADIUS); i++) {

    // Load Z = (2R+i) to registers
    t0_behind1 = t0_current;
    t0_current = t0_infront1;
    t0_infront1 = a[in_index]; // Z = 2R+i   

    in_index += stride;
    next_index += stride;
    
    // Load Z = R+i to shared memory
    if (threadIdx.y < RADIUS) {
      ds_a[threadIdx.y][tx][1] = a[next_index - (RADIUS * dimx)];
      ds_a[threadIdx.y + BLOCK_DIMY + RADIUS][tx][1] = a[next_index + (dimx * BLOCK_DIMY)];
    }
  
    // Load left/right halo data
    if (threadIdx.x < RADIUS) {
      ds_a[ty][threadIdx.x][1] = a[next_index - RADIUS];
      ds_a[ty][threadIdx.x + BLOCK_DIMX + RADIUS][1] = a[next_index + BLOCK_DIMX];
    }
    ds_a[ty][tx][1] = t0_current;

    __syncthreads();

    // Compute stencil for Z = R+i (t + 1) but exclude ghost zones
    if ( (row >= 2*RADIUS) && (row < (dimy-2*RADIUS)) && (col >= 2*RADIUS) && (col < (dimx-2*RADIUS)) && (i < dimz-5*RADIUS) ) {
      t1_infront1 = coeff[0] * t0_current +
	coeff[1] * ds_a[ty][tx-1][1] +
	coeff[2] * ds_a[ty][tx+1][1] +
	coeff[3] * ds_a[ty-1][tx][1] +
	coeff[4] * ds_a[ty+1][tx][1] +
	coeff[5] * t0_behind1 +
	coeff[6] * t0_infront1;
    } else {
      t1_infront1 = t0_current;
    }

    __syncthreads();

    // Load Z = k (t + 1) to shared memory
    ds_a[ty][tx][0] = t1_current;

    __syncthreads();
    
    // Compute stencil for Z = k (t + 2) but exclude halo zones
    if ( (threadIdx.y >= RADIUS) && (threadIdx.y < (BLOCK_DIMY-RADIUS)) && (threadIdx.x >= RADIUS) && (threadIdx.x < (BLOCK_DIMX-RADIUS)) ) {    
      b[out_index] = coeff[0] * t1_current +
	coeff[1] * ds_a[ty][tx-1][0] +
	coeff[2] * ds_a[ty][tx+1][0] +
	coeff[3] * ds_a[ty-1][tx][0] +
	coeff[4] * ds_a[ty+1][tx][0] +
	coeff[5] * t1_behind1 +
	coeff[6] * t1_infront1;
    }

    out_index += stride;
    t1_behind1 = t1_current;
    t1_current = t1_infront1;
  }

}

/* 
   Optimization o1_o2_o5: for iteration on Z axis + use of shared memory + pragma unroll
*/
__global__ void calc_stencil_o1_o2_o5(float *a, float *b, int dimx, int dimy, int dimz, clock_t *runtime) {

  // Shared memory declaration
  __shared__ float ds_a[BLOCK_DIMY+2*RADIUS][BLOCK_DIMX+2*RADIUS];

  int tx = threadIdx.x + RADIUS;
  int ty = threadIdx.y + RADIUS;

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
	
  // (depth/row/col + RADIUS) to compensate the halo
  int in_index = (row+RADIUS) * dimx + (col+RADIUS); // Index for reading Z values (+1 to compensate the halo zones)
  int out_index = 0; // Index for writing output
  
  int stride = dimx * dimy; // Distance between 2D slices
  
  register float infront; // Variable to store the value in front (in the Z axis) of the current slice
  register float behind; // Variable to store the value behind (in the Z axis) the current slice
  register float current; // Input value in the current slice

  // register clock_t time_aux1 = 0, time_aux2 = 0, time_aux3 = 0, time_aux4 = 0, time_aux5 = 0;

  // Load initial values (behind will be loaded inside the next 'for')
  current = a[in_index];
  out_index = in_index;
  in_index += stride;
  
  infront = a[in_index];
  in_index += stride;

  // Iterate over the Z axis
  #pragma unroll
  for (int i = RADIUS; i < dimz - RADIUS; i++) {

    // Load the new values in Z axis
    behind = current;
    current = infront;
    infront = a[in_index];

    in_index += stride;
    out_index += stride;

    // if ( (row == RADIUS) && (col == RADIUS) ) { 
    //   time_aux1 = clock64();
    // }

    // Load above/below halo data to shared memory
    if (threadIdx.y < RADIUS) {
      ds_a[threadIdx.y][tx] = a[out_index - (RADIUS * dimx)];
      ds_a[threadIdx.y + BLOCK_DIMY + RADIUS][tx] = a[out_index + (dimx * BLOCK_DIMY)];
    }

    // Load left/right halo data to shared memory
    if (threadIdx.x < RADIUS) {
      ds_a[ty][threadIdx.x] = a[out_index - RADIUS];
      ds_a[ty][threadIdx.x + BLOCK_DIMX + RADIUS] = a[out_index + BLOCK_DIMX];
    }

    // Load current position to shared memory
    ds_a[ty][tx] = current;

    __syncthreads();

    // if ( (row == RADIUS) && (col == RADIUS) ) {
    //   time_aux2 = clock64();
    //   time_aux4 += time_aux2 - time_aux1;
    // }

    // Compute stencil (7 single precision mul + 6 single precision add)
    b[out_index] = coeff[0] * current +
      coeff[1] * ds_a[ty][tx-1] +
      coeff[2] * ds_a[ty][tx+1] +
      coeff[3] * ds_a[ty-1][tx] +
      coeff[4] * ds_a[ty+1][tx] +
      coeff[5] * behind +
      coeff[6] * infront;
  
    // if ( (row == RADIUS) && (col == RADIUS) ) {
    //   time_aux3 = clock64();
    //   time_aux5 += time_aux3 - time_aux2;
    // }
  }
  // if ( (row == RADIUS) && (col == RADIUS) ) {
  //   runtime[0] = time_aux4;
  //   runtime[1] = time_aux5;
  // }
}

/* 
   Optimization o7: use of read only cache (texture memory)
*/
__global__ void calc_stencil_o7(const float* __restrict__ a, float* __restrict__ b, int dimx, int dimy) {

  int tx = threadIdx.x;
  int ty = threadIdx.y;
  int tz = threadIdx.z;
	
  int row = blockIdx.y * blockDim.y + ty;
  int col = blockIdx.x * blockDim.x + tx;
  int depth = blockIdx.z * blockDim.z + tz;
	
  // (depth/row/col + 1) to compensate the halo
  int index = (depth+1) * dimx * dimy + (row+1) * dimx + (col+1);
  
  // Compute stencil
  b[index] = coeff[0] * __ldg(&a[index]) +
    coeff[1] * __ldg(&a[index-1]) +
    coeff[2] * __ldg(&a[index+1]) +
    coeff[3] * __ldg(&a[index-dimx]) +
    coeff[4] * __ldg(&a[index+dimx]) +
    coeff[5] * __ldg(&a[index+(dimx*dimy)]) +
    coeff[6] * __ldg(&a[index-(dimx*dimy)]);
}

/* 
   Optimization o1_o7: use of shared memory + read only cache (texture memory)
*/
__global__ void calc_stencil_o1_o7(const float* __restrict__ a, float* __restrict__ b, int dimx, int dimy, clock_t *runtime) {

  // Shared Memory Declaration
  __shared__ float ds_a[BLOCK_DIMY+2*RADIUS][BLOCK_DIMX+2*RADIUS];

  int tx = threadIdx.x + RADIUS;
  int ty = threadIdx.y + RADIUS;
  int tz = threadIdx.z + RADIUS;
	
  int row = blockIdx.y * blockDim.y + ty;
  int col = blockIdx.x * blockDim.x + tx;
  int depth = blockIdx.z * blockDim.z + tz;
  
  int index = (depth) * dimx * dimy + (row) * dimx + (col);

  int stride = dimx * dimy; // Distance between 2D slices

  // Measure clock() init of every thread
  // runtime[(depth-RADIUS) * (dimx-2*RADIUS) * (dimy-2*RADIUS) + (row-RADIUS) * (dimx-2*RADIUS) + (col-RADIUS)] = clock64();

  // read_clk_cycles(runtime[0],RADIUS);

  // Load above/below halo data to shared memory
  if (threadIdx.y < RADIUS) {
    ds_a[threadIdx.y][tx] = __ldg(&a[index-(RADIUS*dimx)]);
    ds_a[threadIdx.y + BLOCK_DIMY + RADIUS][tx] = __ldg(&a[index+(BLOCK_DIMY*dimx)]);
  }

  // Load left/right halo data to shared memory
  if (threadIdx.x < RADIUS) {
    ds_a[ty][threadIdx.x] = __ldg(&a[index-RADIUS]);
    ds_a[ty][threadIdx.x + BLOCK_DIMX + RADIUS] = __ldg(&a[index+BLOCK_DIMX]);
  }

  // Load current position to shared memory
  ds_a[ty][tx] = __ldg(&a[index]);

  __syncthreads();

  // read_clk_cycles(runtime[1],RADIUS);

  // Compute stencil
  b[index] = coeff[0] * ds_a[ty][tx] +
    coeff[1] * ds_a[ty][tx-1] +
    coeff[2] * ds_a[ty][tx+1] +
    coeff[3] * ds_a[ty-1][tx] +
    coeff[4] * ds_a[ty+1][tx] +
    coeff[5] * __ldg(&a[index-stride]) +
    coeff[6] * __ldg(&a[index+stride]);

  // read_clk_cycles(runtime[2],RADIUS);
}

/* 
   Optimization o2_o7: use of iteration on Z axis + read only cache (texture memory)
*/

__global__ void calc_stencil_o2_o7(const float* __restrict__ a, float* __restrict__ b, int dimx, int dimy, int dimz) {

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
	
  // (depth/row/col + RADIUS) to compensate the halo
  int in_index = (row+RADIUS) * dimx + (col+RADIUS); // Index for reading Z values (+1 to compensate the halo zones)
  int out_index = 0; // Index for writing output
  
  int stride = dimx * dimy; // Distance between 2D slices
  
  register float infront; // Variable to store the value in front (in the Z axis) of the current slice
  register float behind; // Variable to store the value behind (in the Z axis) the current slice
  register float current; // Input value in the current slice

  // Load initial values (behind will be loaded inside the next 'for')
  current = __ldg(&a[in_index]);
  out_index = in_index;
  in_index += stride;
  
  infront = __ldg(&a[in_index]);
  in_index += stride;

  // Iterate over the Z axis
  for (int i = RADIUS; i < dimz - RADIUS; i++) {

    // Load the new values in Z axis
    behind = current;
    current = infront;
    infront = __ldg(&a[in_index]);

    in_index += stride;
    out_index += stride;

    // Compute stencil
    b[out_index] = coeff[0] * current +
      coeff[1] * __ldg(&a[out_index-1]) +
      coeff[2] * __ldg(&a[out_index+1]) +
      coeff[3] * __ldg(&a[out_index-dimx]) +
      coeff[4] * __ldg(&a[out_index+dimx]) +
      coeff[5] * behind +
      coeff[6] * infront;
  }

}

/* 
   Optimization o3_o7: use of iteration on Z axis without registers + read only cache (texture memory)
*/

__global__ void calc_stencil_o3_o7(const float* __restrict__ a, float* __restrict__ b, int dimx, int dimy, int dimz) {

  int row = blockIdx.y * blockDim.y + threadIdx.y;
  int col = blockIdx.x * blockDim.x + threadIdx.x;
	
  // (depth/row/col + 1) to compensate the halo
  int out_index = (row+RADIUS) * dimx + (col+RADIUS); // Index for writing output
  
  int stride = dimx * dimy; // Distance between 2D slices
  
  // Iterate over the Z axis
  for (int i = RADIUS; i < dimz - RADIUS; i++) {

    out_index += stride;

    // Compute stencil
    b[out_index] = coeff[0] * __ldg(&a[out_index]) +
      coeff[1] * __ldg(&a[out_index-1]) +
      coeff[2] * __ldg(&a[out_index+1]) +
      coeff[3] * __ldg(&a[out_index-dimx]) +
      coeff[4] * __ldg(&a[out_index+dimx]) +
      coeff[5] * __ldg(&a[out_index-stride]) +
      coeff[6] * __ldg(&a[out_index+stride]);
  }

}

/* 
   Optimization o1_o2_o4_o7: shared memory + for iteration on Z axis + temporal blocking (will always compute 2 time iterations)
*/
__global__ void calc_stencil_o1_o2_o4_o7(const float* __restrict__ a, float* __restrict__ b, int dimx, int dimy, int dimz) {

  // Shared memory declaration
  __shared__ float ds_a[BLOCK_DIMY][BLOCK_DIMX];

  int row = blockIdx.y * (BLOCK_DIMY-2*RADIUS) + threadIdx.y + RADIUS;
  int col = blockIdx.x * (BLOCK_DIMX-2*RADIUS) + threadIdx.x + RADIUS;
	
  int in_index = row * dimx + col; // Index for reading Z values
  int out_index = 0; // Index for writing output
  int next_index = 0; // Index for plane Z = output + RADIUS
  
  int stride = dimx * dimy; // Distance between 2D slices

  // t0 = t + 0
  register float t0_infront1; // Variable to store the value ahead (in the Z axis) of the current slice
  register float t0_behind1; // Variable to store the value behind (in the Z axis) the current slice
  register float t0_current; // Input value in the current slice

  // t1 = t + 1
  register float t1_infront1; // Variable to store the value ahead (in the Z axis) of the current slice
  register float t1_behind1; // Variable to store the value behind (in the Z axis) the current slice
  register float t1_current; // Value in current slice for t+1

  // Load ghost zones
  in_index += stride;
  t0_behind1 = __ldg(&a[in_index]); // Z = -R
  in_index += stride;
  next_index = in_index + ((RADIUS-1)*stride); // Z = R-1
  
  out_index = in_index; // Index for writing output, Z = 0
  
  // Load current to (2RADIUS-1) planes
  t0_current = __ldg(&a[in_index]); // Z = 0
  in_index += stride;
  t0_infront1 = __ldg(&a[in_index]); // Z = (2R-1) = 1
  in_index += stride;

  // Compute stencil for Z = 0 (t + 1) but exclude ghost zones 
   if ( (row >= 2*RADIUS) && (row < (dimy-2*RADIUS)) && (col >= 2*RADIUS) && (col < (dimx-2*RADIUS)) ) {
    t1_current = coeff[0] * t0_current +
      coeff[1] * __ldg(&a[out_index-1]) +
      coeff[2] * __ldg(&a[out_index+1]) +
      coeff[3] * __ldg(&a[out_index-dimx]) +
      coeff[4] * __ldg(&a[out_index+dimx]) +
      coeff[5] * t0_behind1 +
      coeff[6] * t0_infront1;
   } else {
     t1_current = t0_current;
   }
  
  // Copy planes Z = -1 to -R to registers in t+1 (ghost zones, keep values in 0.0)
  t1_behind1 = t0_behind1;
  
  for (int i = 0; i < dimz-(4*RADIUS); i++) {
    // Load Z = (2R+i) to registers
    t0_behind1 = t0_current;
    t0_current = t0_infront1;
    t0_infront1 = __ldg(&a[in_index]); // Z = 2R+i   

    in_index += stride;
    next_index += stride;
    
    // Compute stencil for Z = R+i (t + 1) but exclude ghost zones
    if ( (row >= 2*RADIUS) && (row < (dimy-2*RADIUS)) && (col >= 2*RADIUS) && (col < (dimx-2*RADIUS)) && (i < dimz-5*RADIUS) ) {
      t1_infront1 = coeff[0] * t0_current +
	coeff[1] * __ldg(&a[next_index-1]) +
	coeff[2] * __ldg(&a[next_index+1]) +
	coeff[3] * __ldg(&a[next_index-dimx]) +
	coeff[4] * __ldg(&a[next_index+dimx]) +
	coeff[5] * t0_behind1 +
	coeff[6] * t0_infront1;
    } else {
      t1_infront1 = t0_current;
    }

    __syncthreads();

    // Load Z = k (t + 1) to shared memory
    ds_a[threadIdx.y][threadIdx.x] = t1_current;
    
    __syncthreads();

    // Compute stencil for Z = k (t + 2) but exclude halo zones
    if ( (threadIdx.y >= RADIUS) && (threadIdx.y < (BLOCK_DIMY-RADIUS)) && (threadIdx.x >= RADIUS) && (threadIdx.x < (BLOCK_DIMX-RADIUS)) ) {    
      b[out_index] = coeff[0] * t1_current +
	coeff[1] * ds_a[threadIdx.y][threadIdx.x-1] +
	coeff[2] * ds_a[threadIdx.y][threadIdx.x+1] +
	coeff[3] * ds_a[threadIdx.y-1][threadIdx.x] +
	coeff[4] * ds_a[threadIdx.y+1][threadIdx.x] +
	coeff[5] * t1_behind1 +
	coeff[6] * t1_infront1;
    }

    out_index += stride;
    t1_behind1 = t1_current;
    t1_current = t1_infront1;
  }
  
}

void initGold(float *a, int dimx, int dimy, int dimz) {

  for (int i = 0; i < dimz; i++) {
    for (int j = 0; j < dimy; j++) {
      for (int k = 0; k < dimx; k++) {
	if (i<RADIUS || j<RADIUS || i>=dimz-RADIUS || j>=dimy-RADIUS || k<RADIUS || k>=dimx-RADIUS) {
	  a[i*dimx*dimy + j*dimx + k] = 0.0;
        } else {
	  a[i*dimx*dimy + j*dimx + k] = 1.0;
	}
      }
    }
  }

}

void initGoldTemporal(float *a, int dimx, int dimy, int dimz) {

  for (int i = 0; i < dimz; i++) {
    for (int j = 0; j < dimy; j++) {
      for (int k = 0; k < dimx; k++) {
	if (i<2*RADIUS || j<2*RADIUS || i>=dimz-2*RADIUS || j>=dimy-2*RADIUS || k<2*RADIUS || k>=dimx-2*RADIUS) {
	  a[i*dimx*dimy + j*dimx + k] = 0.0;
        } else {
	  a[i*dimx*dimy + j*dimx + k] = 1.0;
	}
      }
    }
  }

}

void hostStencil(float *a, int t_end, int dimx, int dimy, int dimz, float *hcoeff) {

  float *b;

  b = (float *)malloc(dimx * dimy * dimz * sizeof(float));
  initGold(b, dimx, dimy, dimz);

  for (int t = 0; t < t_end; t++) {
    for (int i = 1; i < dimz-1; i++) {
      for (int j = 1; j < dimy-1; j++) {
	for (int k = 1; k < dimx-1; k++) {
	  int index = i*dimx*dimy + j*dimx + k;
	  if (t%2) {
	    a[index] = hcoeff[0] * b[index] +
	      hcoeff[1] * b[index-1] +
	      hcoeff[2] * b[index+1] +
	      hcoeff[3] * b[index-dimx] +
	      hcoeff[4] * b[index+dimx] +
	      hcoeff[5] * b[index-(dimx*dimy)] +
	      hcoeff[6] * b[index+(dimx*dimy)];
	  } else {
	    b[index] = hcoeff[0] * a[index] +
	      hcoeff[1] * a[index-1] +
	      hcoeff[2] * a[index+1] +
	      hcoeff[3] * a[index-dimx] +
	      hcoeff[4] * a[index+dimx] +
	      hcoeff[5] * a[index-(dimx*dimy)] +
	      hcoeff[6] * a[index+(dimx*dimy)];
	  }
	}
      }
    }
  }  

  if (t_end%2) {
    for (int i = 1; i < dimz-1; i++) {
      for (int j = 1; j < dimy-1; j++) {
	for (int k = 1; k < dimx-1; k++) {
	  a[i*dimx*dimy + j*dimx + k] = b[i*dimx*dimy + j*dimx + k];
	}
      }
    }    
  } 
  free(b);

}

void hostStencilTemporal(float *a, int t_end, int dimx, int dimy, int dimz, float *hcoeff) {

  float *b;

  b = (float *)malloc(dimx * dimy * dimz * sizeof(float));
  initGoldTemporal(b, dimx, dimy, dimz);

  for (int t = 0; t < t_end; t++) {
    for (int i = 2*RADIUS; i < dimz-2*RADIUS; i++) {
      for (int j = 2*RADIUS; j < dimy-2*RADIUS; j++) {
	for (int k = 2*RADIUS; k < dimx-2*RADIUS; k++) {
	  int index = i*dimx*dimy + j*dimx + k;
	  if (t%2) {
	    a[index] = hcoeff[0] * b[index] +
	      hcoeff[1] * b[index-1] +
	      hcoeff[2] * b[index+1] +
	      hcoeff[3] * b[index-dimx] +
	      hcoeff[4] * b[index+dimx] +
	      hcoeff[5] * b[index-(dimx*dimy)] +
	      hcoeff[6] * b[index+(dimx*dimy)];
	  } else {
	    b[index] = hcoeff[0] * a[index] +
	      hcoeff[1] * a[index-1] +
	      hcoeff[2] * a[index+1] +
	      hcoeff[3] * a[index-dimx] +
	      hcoeff[4] * a[index+dimx] +
	      hcoeff[5] * a[index-(dimx*dimy)] +
	      hcoeff[6] * a[index+(dimx*dimy)];
	  }
	}
      }
    }
  }  

  if (t_end%2) {
    for (int i = 2*RADIUS; i < dimz-2*RADIUS; i++) {
      for (int j = 2*RADIUS; j < dimy-2*RADIUS; j++) {
	for (int k = 2*RADIUS; k < dimx-2*RADIUS; k++) {
	  a[i*dimx*dimy + j*dimx + k] = b[i*dimx*dimy + j*dimx + k];
	}
      }
    }    
  } 
  free(b);

}


void printMatrix(float *a, int dimx, int dimy, int dimz) {
 
  for (int k=0; k < dimz; k++) {    
    for (int i=0; i < dimy; i++) {
      for (int j=0; j < dimx; j++) {
	printf("%f, ",a[k*dimx*dimy + i*dimx + j]);
      }
      printf("\n");
    }
    printf("\n");
  }
}

bool checkResult(float *a, float *ref, int dimx, int dimy, int dimz) {

  for (int i = 0; i < dimz; i++) {
    for (int j = 0; j < dimy; j++) {
      for (int k = 0; k < dimx; k++) {
	if (a[i*dimx*dimy + j*dimx + k] != ref[i*dimx*dimy + j*dimx + k]) {
	  printf("Expected: %f, received: %f at position [%d,%d,%d]\n",ref[i*dimx*dimy+j*dimx+k],a[i*dimx*dimy+j*dimx+k],i,j,k);
	  return 0;
	}
      }
    }
  }    

  return 1;

}

int main(int argc, char* argv[]) {

  float *h_a, *h_gold_a;
  float *d_a, *d_b;
  float hcoeff[7] = {1.0,1.0,1.0,1.0,1.0,1.0,1.0}; //cc, cw, ce, cn, cs, ct, cb

  hipEvent_t t0, t1, t2, t3, t4, t5;
  float init, host_comp, host2gpu, gpu2host, gpu_comp, tot;
  int dimx, dimy, dimz, t_end;
  long points, flop;
  float gFlops;
  int opt; // Variable to select the optimization

#ifdef MEASURE_KERNEL_RUNTIME
  clock_t *h_runtime; // Variables to benchmark elapsed time (in clock cycles) inside the kernel
#endif
  clock_t *d_runtime;

  if (argc != 6) {
    printf(" use: <exec> <OPT> <DIMX> <DIMY> <DIMZ> <T_END>\n");
    exit(-1);
  }
  opt = atoi(argv[1]);
  dimx = atoi(argv[2]);
  dimy = atoi(argv[3]);
  dimz = atoi(argv[4]);
  t_end = atoi(argv[5]);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, 0);

  hipEventCreate(&t0);
  hipEventCreate(&t1);
  hipEventCreate(&t2);
  hipEventCreate(&t3);
  hipEventCreate(&t4);
  hipEventCreate(&t5);

  int gold_size;

  // If temporal blocking is requested, allocate more device memory
  if ( (opt == 124) || (opt == 1247) ) {
    gold_size = (dimx+4*RADIUS) * (dimy+4*RADIUS) * (dimz+4*RADIUS) * sizeof(float);
    // Check if the number of iterations is even
    if ( (t_end%2) != 0) {
      printf("Number of time iterations is odd, adding one iteration!\n");
      t_end++;
    }
  } else {
    gold_size = (dimx+2*RADIUS) * (dimy+2*RADIUS) * (dimz+2*RADIUS) * sizeof(float);
  }
  points = (long)dimx * (long)dimy * (long)dimz * (long)t_end;
  flop = (long)(6 + 7) * points; // 6 adds, 7 multiplies

  hipEventRecord(t0);

  /* allocate device variables */
  wbCheck(hipMalloc((void**) &d_a, gold_size));
  wbCheck(hipMalloc((void**) &d_b, gold_size));
  wbCheck(hipMalloc((void**) &d_runtime, 3*sizeof(clock_t)));
  // wbCheck(hipMalloc((void**) &d_runtime, 3*BLOCK_DIMX*BLOCK_DIMY*sizeof(clock_t)));

  /* allocate host variables */
  h_a = (float *)malloc(gold_size);
  h_gold_a = (float *)malloc(gold_size);

#ifdef MEASURE_KERNEL_RUNTIME
  h_runtime = (clock_t *)malloc(3*sizeof(clock_t));
  // h_runtime = (clock_t *)malloc(3*BLOCK_DIMX*BLOCK_DIMY*sizeof(clock_t));
#endif

  if ( (opt == 124) || (opt == 1247) ) {
    initGoldTemporal(h_a, dimx+4*RADIUS, dimy+4*RADIUS, dimz+4*RADIUS);
    initGoldTemporal(h_gold_a, dimx+4*RADIUS, dimy+4*RADIUS, dimz+4*RADIUS);
  } else {
    initGold(h_a, dimx+2*RADIUS, dimy+2*RADIUS, dimz+2*RADIUS);
    initGold(h_gold_a, dimx+2*RADIUS, dimy+2*RADIUS, dimz+2*RADIUS);
  }

  hipEventRecord(t1);

  if ( (opt == 124) || (opt == 1247) ) {
    hostStencilTemporal(h_gold_a, t_end, dimx+4*RADIUS, dimy+4*RADIUS, dimz+4*RADIUS, hcoeff);
  } else {
    hostStencil(h_gold_a, t_end, dimx+2*RADIUS, dimy+2*RADIUS, dimz+2*RADIUS, hcoeff);
  }
  
#ifdef PRINT_GOLD
  if ( (opt == 124) || (opt == 1247) ) {
    printMatrix(h_gold_a,dimx+4*RADIUS, dimy+4*RADIUS, dimz+4*RADIUS);    
  } else {  
    printMatrix(h_gold_a,dimx+2*RADIUS, dimy+2*RADIUS, dimz+2*RADIUS);
  }
#endif

  hipEventRecord(t2);

  wbCheck(hipMemcpyToSymbol(HIP_SYMBOL(coeff), hcoeff, sizeof(hcoeff)));
  wbCheck(hipMemcpy(d_a, h_a, gold_size, hipMemcpyHostToDevice)); // Initialize device values
  wbCheck(hipMemcpy(d_b, d_a, gold_size, hipMemcpyDeviceToDevice)); // Copy contents from d_a to d_b
 
  hipEventRecord(t3);

  dim3 dimBlock;
  dim3 dimGrid;

  switch (opt) {
  case 0:
    printf("Optimization level: o0\n");
    dimBlock.x = BLOCK_DIMX;
    dimBlock.y = BLOCK_DIMY;
    dimBlock.z = BLOCK_DIMZ;
    dimGrid.x = (int)ceil(dimx/BLOCK_DIMX);
    dimGrid.y = (int)ceil(dimy/BLOCK_DIMY);
    dimGrid.z = (int)ceil(dimz/BLOCK_DIMZ);

    for (int i = 0; i < t_end; i++) {
      if (i%2) {
	calc_stencil_o0 <<< dimGrid,dimBlock >>> (d_b, d_a, dimx+2, dimy+2);
      } else {
	calc_stencil_o0 <<< dimGrid,dimBlock >>> (d_a, d_b, dimx+2, dimy+2);
      }
      wbCheck(hipGetLastError());
    }
    break;

  case 1:
    printf("Optimization level: o1\n");
    dimBlock.x = BLOCK_DIMX;
    dimBlock.y = BLOCK_DIMY;
    dimBlock.z = BLOCK_DIMZ;
    dimGrid.x = (int)ceil(dimx/BLOCK_DIMX);
    dimGrid.y = (int)ceil(dimy/BLOCK_DIMY);
    dimGrid.z = (int)ceil(dimz/BLOCK_DIMZ);

    for (int i = 0; i < t_end; i++) {
      if (i%2) {
	calc_stencil_o1 <<< dimGrid,dimBlock >>> (d_b, d_a, dimx+2, dimy+2, d_runtime);
      } else {
	calc_stencil_o1 <<< dimGrid,dimBlock >>> (d_a, d_b, dimx+2, dimy+2, d_runtime);
      }
      wbCheck(hipGetLastError());
    }
    break;

  case 2:
    printf("Optimization level: o2\n");
    dimBlock.x = BLOCK_DIMX;
    dimBlock.y = BLOCK_DIMY;
    dimBlock.z = 1;
    dimGrid.x = (int)ceil(dimx/BLOCK_DIMX);
    dimGrid.y = (int)ceil(dimy/BLOCK_DIMY);
    dimGrid.z = 1;

    for (int i = 0; i < t_end; i++) {
      if (i%2) {
	calc_stencil_o2 <<< dimGrid,dimBlock >>> (d_b, d_a, dimx+2, dimy+2, dimz+2);
      } else {
	calc_stencil_o2 <<< dimGrid,dimBlock >>> (d_a, d_b, dimx+2, dimy+2, dimz+2);
      }
      wbCheck(hipGetLastError());
    }
    break;

  case 3:
    printf("Optimization level: o3\n");
    dimBlock.x = BLOCK_DIMX;
    dimBlock.y = BLOCK_DIMY;
    dimBlock.z = 1;
    dimGrid.x = (int)ceil(dimx/BLOCK_DIMX);
    dimGrid.y = (int)ceil(dimy/BLOCK_DIMY);
    dimGrid.z = 1;

    for (int i = 0; i < t_end; i++) {
      if (i%2) {
	calc_stencil_o3 <<< dimGrid,dimBlock >>> (d_b, d_a, dimx+2, dimy+2, dimz+2);
      } else {
	calc_stencil_o3 <<< dimGrid,dimBlock >>> (d_a, d_b, dimx+2, dimy+2, dimz+2);
      }
      wbCheck(hipGetLastError());
    }
    break;

  case 25:
    printf("Optimization level: o2_o5\n");
    dimBlock.x = BLOCK_DIMX;
    dimBlock.y = BLOCK_DIMY;
    dimBlock.z = 1;
    dimGrid.x = (int)ceil(dimx/BLOCK_DIMX);
    dimGrid.y = (int)ceil(dimy/BLOCK_DIMY);
    dimGrid.z = 1;

    for (int i = 0; i < t_end; i++) {
      if (i%2) {
	calc_stencil_o2_o5 <<< dimGrid,dimBlock >>> (d_b, d_a, dimx+2, dimy+2, dimz+2);
      } else {
	calc_stencil_o2_o5 <<< dimGrid,dimBlock >>> (d_a, d_b, dimx+2, dimy+2, dimz+2);
      }
      wbCheck(hipGetLastError());
    }
    break;

  case 12:
    printf("Optimization level: o1_o2\n");
    dimBlock.x = BLOCK_DIMX;
    dimBlock.y = BLOCK_DIMY;
    dimBlock.z = 1;
    dimGrid.x = (int)ceil(dimx/BLOCK_DIMX);
    dimGrid.y = (int)ceil(dimy/BLOCK_DIMY);
    dimGrid.z = 1;

    for (int i = 0; i < t_end; i++) {
      if (i%2) {
	calc_stencil_o1_o2 <<< dimGrid,dimBlock >>> (d_b, d_a, dimx+2, dimy+2, dimz+2, d_runtime);
      } else {
	calc_stencil_o1_o2 <<< dimGrid,dimBlock >>> (d_a, d_b, dimx+2, dimy+2, dimz+2, d_runtime);
      }
      wbCheck(hipGetLastError());
    }
    break;

  case 13:
    printf("Optimization level: o1_o3\n");
    dimBlock.x = BLOCK_DIMX;
    dimBlock.y = BLOCK_DIMY;
    dimBlock.z = 1;
    dimGrid.x = (int)ceil(dimx/BLOCK_DIMX);
    dimGrid.y = (int)ceil(dimy/BLOCK_DIMY);
    dimGrid.z = 1;

    for (int i = 0; i < t_end; i++) {
      if (i%2) {
	calc_stencil_o1_o3 <<< dimGrid,dimBlock >>> (d_b, d_a, dimx+2, dimy+2, dimz+2, d_runtime);
      } else {
	calc_stencil_o1_o3 <<< dimGrid,dimBlock >>> (d_a, d_b, dimx+2, dimy+2, dimz+2, d_runtime);
      }
      wbCheck(hipGetLastError());
    }
    break;

  case 124:
    printf("Optimization level: o1_o2_o4\n");
    dimBlock.x = BLOCK_DIMX;
    dimBlock.y = BLOCK_DIMY;
    dimBlock.z = 1;
    dimGrid.x = (int)ceil(dimx/(BLOCK_DIMX-2*RADIUS));
    dimGrid.y = (int)ceil(dimy/(BLOCK_DIMY-2*RADIUS));
    dimGrid.z = 1;

    for (int i = 0; i < t_end/2; i++) {
      if (i%2) {
	calc_stencil_o1_o2_o4 <<< dimGrid,dimBlock >>> (d_b, d_a, dimx+4*RADIUS, dimy+4*RADIUS, dimz+4*RADIUS);
      } else {
	calc_stencil_o1_o2_o4 <<< dimGrid,dimBlock >>> (d_a, d_b, dimx+4*RADIUS, dimy+4*RADIUS, dimz+4*RADIUS);
      }
      wbCheck(hipGetLastError());
    }
    break;

  case 125:
    printf("Optimization level: o1_o2_o5\n");
    dimBlock.x = BLOCK_DIMX;
    dimBlock.y = BLOCK_DIMY;
    dimBlock.z = 1;
    dimGrid.x = (int)ceil(dimx/BLOCK_DIMX);
    dimGrid.y = (int)ceil(dimy/BLOCK_DIMY);
    dimGrid.z = 1;

    for (int i = 0; i < t_end; i++) {
      if (i%2) {
	calc_stencil_o1_o2_o5 <<< dimGrid,dimBlock >>> (d_b, d_a, dimx+2, dimy+2, dimz+2, d_runtime);
      } else {
	calc_stencil_o1_o2_o5 <<< dimGrid,dimBlock >>> (d_a, d_b, dimx+2, dimy+2, dimz+2, d_runtime);
      }
      wbCheck(hipGetLastError());
    }
    break;

  case 7:
    printf("Optimization level: o7\n");
    dimBlock.x = BLOCK_DIMX;
    dimBlock.y = BLOCK_DIMY;
    dimBlock.z = BLOCK_DIMZ;
    dimGrid.x = (int)ceil(dimx/BLOCK_DIMX);
    dimGrid.y = (int)ceil(dimy/BLOCK_DIMY);
    dimGrid.z = (int)ceil(dimz/BLOCK_DIMZ);

    for (int i = 0; i < t_end; i++) {
      if (i%2) {
	calc_stencil_o7 <<< dimGrid,dimBlock >>> (d_b, d_a, dimx+2, dimy+2);
      } else {
	calc_stencil_o7 <<< dimGrid,dimBlock >>> (d_a, d_b, dimx+2, dimy+2);
      }
      wbCheck(hipGetLastError());
    }
    break;

  case 17:
    printf("Optimization level: o1_o7\n");
    dimBlock.x = BLOCK_DIMX;
    dimBlock.y = BLOCK_DIMY;
    dimBlock.z = BLOCK_DIMZ;
    dimGrid.x = (int)ceil(dimx/BLOCK_DIMX);
    dimGrid.y = (int)ceil(dimy/BLOCK_DIMY);
    dimGrid.z = (int)ceil(dimz/BLOCK_DIMZ);

    for (int i = 0; i < t_end; i++) {
      if (i%2) {
	calc_stencil_o1_o7 <<< dimGrid,dimBlock >>> (d_b, d_a, dimx+2, dimy+2, d_runtime);
      } else {
	calc_stencil_o1_o7 <<< dimGrid,dimBlock >>> (d_a, d_b, dimx+2, dimy+2, d_runtime);
      }
      wbCheck(hipGetLastError());
    }
    break;

  case 27:
    printf("Optimization level: o2_o7\n");
    dimBlock.x = BLOCK_DIMX;
    dimBlock.y = BLOCK_DIMY;
    dimBlock.z = 1;
    dimGrid.x = (int)ceil(dimx/BLOCK_DIMX);
    dimGrid.y = (int)ceil(dimy/BLOCK_DIMY);
    dimGrid.z = 1;

    for (int i = 0; i < t_end; i++) {
      if (i%2) {
	calc_stencil_o2_o7 <<< dimGrid,dimBlock >>> (d_b, d_a, dimx+2, dimy+2, dimz+2);
      } else {
	calc_stencil_o2_o7 <<< dimGrid,dimBlock >>> (d_a, d_b, dimx+2, dimy+2, dimz+2);
      }
      wbCheck(hipGetLastError());
    }
    break;    

  case 37:
    printf("Optimization level: o3_o7\n");
    dimBlock.x = BLOCK_DIMX;
    dimBlock.y = BLOCK_DIMY;
    dimBlock.z = 1;
    dimGrid.x = (int)ceil(dimx/BLOCK_DIMX);
    dimGrid.y = (int)ceil(dimy/BLOCK_DIMY);
    dimGrid.z = 1;

    for (int i = 0; i < t_end; i++) {
      if (i%2) {
	calc_stencil_o3_o7 <<< dimGrid,dimBlock >>> (d_b, d_a, dimx+2, dimy+2, dimz+2);
      } else {
	calc_stencil_o3_o7 <<< dimGrid,dimBlock >>> (d_a, d_b, dimx+2, dimy+2, dimz+2);
      }
      wbCheck(hipGetLastError());
    }
    break;    

  case 1247:
    printf("Optimization level: o1_o2_o4_o7\n");
    dimBlock.x = BLOCK_DIMX;
    dimBlock.y = BLOCK_DIMY;
    dimBlock.z = 1;
    dimGrid.x = (int)ceil(dimx/(BLOCK_DIMX-2*RADIUS));
    dimGrid.y = (int)ceil(dimy/(BLOCK_DIMY-2*RADIUS));
    dimGrid.z = 1;

    for (int i = 0; i < t_end/2; i++) {
      if (i%2) {
	calc_stencil_o1_o2_o4_o7 <<< dimGrid,dimBlock >>> (d_b, d_a, dimx+4*RADIUS, dimy+4*RADIUS, dimz+4*RADIUS);
      } else {
	calc_stencil_o1_o2_o4_o7 <<< dimGrid,dimBlock >>> (d_a, d_b, dimx+4*RADIUS, dimy+4*RADIUS, dimz+4*RADIUS);
      }
      wbCheck(hipGetLastError());
    }
    break;
    
  default:
    printf("Invalid optimization selected\n");
    break;
  }

  hipEventRecord(t4);
  hipDeviceSynchronize();

  if ( (opt == 124) || (opt == 1247) ) {
    if ((t_end/2)%2) {
      wbCheck(hipMemcpy(h_a, d_b, gold_size, hipMemcpyDeviceToHost));
    } else {
      wbCheck(hipMemcpy(h_a, d_a, gold_size, hipMemcpyDeviceToHost));
    }
  } else {
    if (t_end%2) {
      wbCheck(hipMemcpy(h_a, d_b, gold_size, hipMemcpyDeviceToHost));
    } else {
      wbCheck(hipMemcpy(h_a, d_a, gold_size, hipMemcpyDeviceToHost));
    }
  }
  
  hipEventRecord(t5);

#ifdef MEASURE_KERNEL_RUNTIME
  wbCheck(hipMemcpy(h_runtime, d_runtime, 3*sizeof(clock_t), hipMemcpyDeviceToHost));
  printf("First run time: %f ms\n",((float)(h_runtime[1]-h_runtime[0])/(float)prop.clockRate));
  printf("Second run time: %f ms\n",((float)(h_runtime[2]-h_runtime[1])/(float)prop.clockRate));
#endif

  // wbCheck(hipMemcpy(h_runtime, d_runtime, 3*sizeof(clock_t), hipMemcpyDeviceToHost));
  // printf("Shared memory run time: %f ms\n",((float)(h_runtime[0])/(float)prop.clockRate));
  // printf("Computation run time: %f ms\n",((float)(h_runtime[1])/(float)prop.clockRate));

  // Measure init clock() for every thread
  // wbCheck(hipMemcpy(h_runtime, d_runtime, 3*BLOCK_DIMX*BLOCK_DIMY*sizeof(clock_t), hipMemcpyDeviceToHost));
  // for (int i = 0; i < 3*BLOCK_DIMX*BLOCK_DIMY; i++) printf("runtime[%d] = \t \t %d\n",i,h_runtime[i]);

  hipFree(d_a);
  hipFree(d_b);
 
#ifdef PRINT_RESULT
  if ( (opt == 124) || (opt == 1247) ) {
    printMatrix(h_a,dimx+4*RADIUS,dimy+4*RADIUS,dimz+4*RADIUS);
  } else {
    printMatrix(h_a,dimx+2*RADIUS,dimy+2*RADIUS,dimz+2*RADIUS);
  }
#endif

  if ( (opt == 124) || (opt == 1247) ) {
    if (checkResult(h_a,h_gold_a,dimx+4*RADIUS,dimy+4*RADIUS,dimz+4*RADIUS)) {
      printf("Correct results!\n");
    } else {
      printf("Wrong results!!!!!!\n");
    }
  } else {
    if (checkResult(h_a,h_gold_a,dimx+2*RADIUS,dimy+2*RADIUS,dimz+2*RADIUS)) {
      printf("Correct results!\n");
    } else {
      printf("Wrong results!!!!!!\n");
    }
  }
  
  hipEventSynchronize(t5);

  hipEventElapsedTime(&init, t0, t1);
  hipEventElapsedTime(&host_comp, t1, t2);
  hipEventElapsedTime(&host2gpu, t2, t3);
  hipEventElapsedTime(&gpu_comp, t3, t4);
  hipEventElapsedTime(&gpu2host, t4, t5);
  hipEventElapsedTime(&tot, t0, t5);

  gFlops = (1.0e-6)*flop/gpu_comp;
  
  printf("GPU Clock: %d MHz\n",prop.clockRate/1000);
  printf("DIM = %dx%dx%d; T_END = %d; BLOCK_WIDTH = %dx%dx%d\n", dimx,dimy,dimz,t_end,BLOCK_DIMX,BLOCK_DIMY,BLOCK_DIMZ);
  printf("init=%f, host_comp=%f, host2gpu=%f, gpu_comp=%f, gpu2host=%f, tot=%f \n", 
	 init, host_comp, host2gpu, gpu_comp, gpu2host, tot);
  printf("Stencil Throughput: %f Gpts/s\n", (1.0e-6*points)/gpu_comp); // gpu_comp is measured in ms
  printf("gFlops = %f GFLOPs\n", gFlops);

  free(h_a);
  free(h_gold_a);

  printf("\n");
  return 0;
}
