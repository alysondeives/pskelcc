#include "hip/hip_runtime.h"
#include "jacobi3d7pt_kernel.hu"
__global__ void kernel0(float *A, float *B, int n, int tsteps, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.z, t1 = threadIdx.y, t2 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    for (int c1 = 32 * b0; c1 < n - 1; c1 += 8192)
      if (n >= t0 + c1 + 2 && t0 + c1 >= 1)
        for (int c2 = 32 * b1; c2 < n - 1; c2 += 8192)
          for (int c3 = 0; c3 < n - 1; c3 += 32)
            for (int c5 = ppcg_max(t1, ((t1 + c2 + 3) % 4) - c2 + 1); c5 <= ppcg_min(31, n - c2 - 2); c5 += 4)
              for (int c6 = ppcg_max(t2, ((t2 + c3 + 3) % 4) - c3 + 1); c6 <= ppcg_min(31, n - c3 - 2); c6 += 4)
                B[((t0 + c1) * n + (c2 + c5)) * n + (c3 + c6)] = ((((0.125F * ((A[((t0 + c1 + 1) * n + (c2 + c5)) * n + (c3 + c6)] + (2.F * A[((t0 + c1) * n + (c2 + c5)) * n + (c3 + c6)])) + A[((t0 + c1 - 1) * n + (c2 + c5)) * n + (c3 + c6)])) + (0.125F * ((A[((t0 + c1) * n + (c2 + c5 + 1)) * n + (c3 + c6)] + (2.F * A[((t0 + c1) * n + (c2 + c5)) * n + (c3 + c6)])) + A[((t0 + c1) * n + (c2 + c5 - 1)) * n + (c3 + c6)]))) + (0.125F * ((A[((t0 + c1) * n + (c2 + c5)) * n + (c3 + c6 + 1)] + (2.F * A[((t0 + c1) * n + (c2 + c5)) * n + (c3 + c6)])) + A[((t0 + c1) * n + (c2 + c5)) * n + (c3 + c6 - 1)]))) + (0.125F * A[((t0 + c1) * n + (c2 + c5)) * n + (c3 + c6)]));
}
__global__ void kernel1(float *A, float *B, int n, int tsteps, int c0)
{
    int b0 = blockIdx.y, b1 = blockIdx.x;
    int t0 = threadIdx.z, t1 = threadIdx.y, t2 = threadIdx.x;

    #define ppcg_min(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x < _y ? _x : _y; })
    #define ppcg_max(x,y)    ({ __typeof__(x) _x = (x); __typeof__(y) _y = (y); _x > _y ? _x : _y; })
    for (int c1 = 32 * b0; c1 < n - 1; c1 += 8192)
      if (n >= t0 + c1 + 2 && t0 + c1 >= 1)
        for (int c2 = 32 * b1; c2 < n - 1; c2 += 8192)
          for (int c3 = 0; c3 < n - 1; c3 += 32)
            for (int c5 = ppcg_max(t1, ((t1 + c2 + 3) % 4) - c2 + 1); c5 <= ppcg_min(31, n - c2 - 2); c5 += 4)
              for (int c6 = ppcg_max(t2, ((t2 + c3 + 3) % 4) - c3 + 1); c6 <= ppcg_min(31, n - c3 - 2); c6 += 4)
                A[((t0 + c1) * n + (c2 + c5)) * n + (c3 + c6)] = ((((0.125F * ((B[((t0 + c1 + 1) * n + (c2 + c5)) * n + (c3 + c6)] - (2.F * B[((t0 + c1) * n + (c2 + c5)) * n + (c3 + c6)])) + B[((t0 + c1 - 1) * n + (c2 + c5)) * n + (c3 + c6)])) + (0.125F * ((B[((t0 + c1) * n + (c2 + c5 + 1)) * n + (c3 + c6)] - (2.F * B[((t0 + c1) * n + (c2 + c5)) * n + (c3 + c6)])) + B[((t0 + c1) * n + (c2 + c5 - 1)) * n + (c3 + c6)]))) + (0.125F * ((B[((t0 + c1) * n + (c2 + c5)) * n + (c3 + c6 + 1)] - (2.F * B[((t0 + c1) * n + (c2 + c5)) * n + (c3 + c6)])) + B[((t0 + c1) * n + (c2 + c5)) * n + (c3 + c6 - 1)]))) + (0.125F * B[((t0 + c1) * n + (c2 + c5)) * n + (c3 + c6)]));
}
